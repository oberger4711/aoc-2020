
#include <hip/hip_runtime.h>
#include <iostream>

constexpr int LEN_INITAL = 8;
constexpr int STEPS = 6;

constexpr int LEN_W = 1 + 2 * STEPS;
constexpr int LEN_Z = LEN_W;
constexpr int LEN_Y = LEN_INITAL + 2 * STEPS;
constexpr int LEN_X = LEN_INITAL + 2 * STEPS;
constexpr int LEN_TOTAL = LEN_W * LEN_Z * LEN_Y * LEN_X;

constexpr int SIZE_W = LEN_Z * LEN_Y * LEN_X;
constexpr int SIZE_Z = LEN_Y * LEN_X;
constexpr int SIZE_Y = LEN_X;

constexpr int NUM_THREADS = 512;
constexpr int NUM_BLOCKS = LEN_TOTAL / NUM_THREADS;

inline int coord_to_idx(const int w, const int z, const int y, const int x) {
  return
    w * SIZE_W +
    z * SIZE_Z +
    y * SIZE_Y +
    x;
}

inline void print_slice(const int* grid, const int w, const int z) {
  for (int row = 0; row < LEN_Y; ++row) {
    for (int col = 0; col < LEN_X; ++col) {
      std::cout << grid[coord_to_idx(w, z, row, col)] << " ";
    }
    std::cout << "\n";
  }
}

__device__
int coord_to_idx_dev(const int w, const int z, const int y, const int x) {
  return w * SIZE_W + z * SIZE_Z + y * SIZE_Y + x;
}

__global__
void step(const int* grid, int* grid_next) {
  // Find out where we are.
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < LEN_TOTAL) {
    int left = idx;
    int w = idx / SIZE_W;
    left = idx - w * SIZE_W;
    int z = left / SIZE_Z;
    left = left - z * SIZE_Z;
    int y = left / SIZE_Y;
    int x = left - y * SIZE_Y;

    // TODO: for loop here?
    const int active = grid[idx];
    // Count active neighbors.
    int active_neighbors = 0;
    int min_nw = max(0, w - 1);
    int max_nw = min(LEN_W, w + 2);
    int min_nz = max(0, z - 1);
    int max_nz = min(LEN_Z, z + 2);
    int min_ny = max(0, y - 1);
    int max_ny = min(LEN_Y, y + 2);
    int min_nx = max(0, x - 1);
    int max_nx = min(LEN_X, x + 2);
    for (int nw = min_nw; nw < max_nw; ++nw) {
      for (int nz = min_nz; nz < max_nz; ++nz) {
        for (int ny = min_ny; ny < max_ny; ++ny) {
          for (int nx = min_nx; nx < max_nx; ++nx) {
            active_neighbors += grid[coord_to_idx_dev(nw, nz, ny, nx)];
          }
        }
      }
    }
    active_neighbors -= active;
    // Rules
    int active_next = active;
    if (active == 1 && (active_neighbors < 2 || active_neighbors > 3)) {
      active_next = 0;
    }
    else if (active == 0 && active_neighbors == 3) {
      active_next = 1;
    }
    //active_next = idx;
    grid_next[idx] = active_next;
  }
}

int main() {

  // Initialize grid.
  int* grid;
  hipMallocManaged(&grid, LEN_TOTAL * sizeof(int));
  hipMemset(grid, 0, LEN_TOTAL);
  int initial_grid[LEN_INITAL][LEN_INITAL] = {
    {1, 1, 0, 0, 1, 0, 1, 0},
    {1, 1, 1, 0, 1, 0, 1, 1},
    {0, 0, 1, 1, 1, 0, 0, 1},
    {0, 1, 0, 0, 0, 0, 1, 1},
    {0, 1, 0, 0, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 0, 0, 0},
    {1, 1, 1, 1, 1, 1, 1, 0},
    {1, 0, 1, 1, 0, 1, 0, 1}
  };
  for (int row = 0; row < LEN_INITAL; ++row) {
    for (int col = 0; col < LEN_INITAL; ++col) {
      grid[coord_to_idx(STEPS, STEPS, STEPS + row, STEPS + col)] = initial_grid[row][col];
    }
  }
  //print_slice(grid, STEPS, STEPS);
  int* grid_next;
  hipMallocManaged(&grid_next, LEN_TOTAL * sizeof(int));
  for (int i = 0; i < STEPS; ++i) {
    //std::cout << "Step " << i << "\n";
    step<<<NUM_BLOCKS, NUM_THREADS>>>(grid, grid_next);
    hipDeviceSynchronize();
    std::swap(grid, grid_next);
    //print_slice(grid, STEPS, STEPS);
  }
  // Count actives.
  int count = 0;
  for (int i = 0; i < LEN_TOTAL; ++i) {
    count += grid[i];
  }
  std::cout << "Active: " << count << "\n";
  hipFree(grid);
  hipFree(grid_next);
  return 0;
}
